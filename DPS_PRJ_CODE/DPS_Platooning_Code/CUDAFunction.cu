#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include "" 

#define MINIMUN_SPACING 3
#define MAX_SIZE 10

__global__ void positionCalculateExecution(float* coordinate, float* distance, 
										   float relativePos[MAX_SIZE][2], bool* distanceCheck)
{
	const int tid = threadIdx.x;

	//calculate relative distance
	if (tid == 0)
		relativePos[tid][0] = distance[0];
	else
	{
		for (int i = tid; i >= 0; i--)
		{
			relativePos[tid][0] = distance[tid];
		}
	}
	// check reasonability of distance
	if (distance[tid] > MINIMUN_SPACING)
		distanceCheck[tid] = true;
	else
		distanceCheck[tid] = false;

}


//coordinate[2]:		coordinate of leading truck.two elements(x,y)
//distance[size]:		distance with front truck sorted by following sequence
//relativePos[size][2]:	relative distance based on leading truck
//size:					number of following truck
//distanceCheck:		check distance is reasonanle or not.true:reasonable false:not reasonable
extern "C" void PositionCalculation(float* coordinate, float* distance, float relativePos[MAX_SIZE][2], bool *distanceCheck, int size)
{
	float* dev_coordinate;
	float* dev_dis;
	float dev_rPos[MAX_SIZE][2];
	bool* dev_check;

	//allocate GPU memory
	hipMalloc((void**)&dev_coordinate, 2 * sizeof(float));
	hipMalloc((void**)&dev_dis, size * sizeof(float));
	hipMalloc((void**)&dev_rPos, 2 * size * sizeof(float));
	hipMalloc((void**)&dev_check, size * sizeof(bool));

	//Copy input arry from host memory to GPU buffers. 
	hipMemcpy(dev_coordinate, coordinate, 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_dis, coordinate, size * sizeof(float), hipMemcpyHostToDevice);

	//call kernel function
	positionCalculateExecution << <1, size >> > (dev_coordinate, dev_dis, dev_rPos, dev_check);

	//Copy output arry from GPU device to Host
	hipMemcpy(relativePos, dev_rPos, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(distanceCheck, dev_check, size * sizeof(float), hipMemcpyDeviceToHost);

	//Free device memory
	hipFree(dev_coordinate);
	hipFree(dev_dis);
	hipFree(dev_rPos);
	hipFree(dev_check);
}


