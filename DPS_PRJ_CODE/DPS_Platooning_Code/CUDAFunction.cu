#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include "" 

__global__ void positionCalculateExecution(float* coordinate, float* distance, float* relativePos)
{
	const int tid = threadIdx.x;

	if (tid == 0)
		relativePos[tid] = distance[0];

	else
	{
		for (int i = tid; i >= 0; i--)
		{
			relativePos[tid] = distance[tid];
		}
	}
}


//coordinate:	coordinate of leading truck.two elements(x,y)
//distance:		distance with front truck sorted by following sequence
//relativeDis:	relative distance based on leading truck
//size:			number of following truck
extern "C" void PositionCalculation(float* coordinate, float* distance, float* relativePos, int size)
{
	float* dev_coordinate;
	float* dev_dis;
	float* dev_rPos;

	//allocate GPU memory
	hipMalloc((void**)&dev_coordinate, 2 * sizeof(float));
	//hipMalloc((void**)&dev_dis, size * sizeof(float));
	positionCalculateExecution << <1, size >> > (dev_coordinate, dev_dis, dev_rPos);
	hipMalloc((void**)&dev_rPos, size * sizeof(float));

	//Copy input arry from host memory to GPU buffers. 
	hipMemcpy(dev_coordinate, coordinate, 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_dis, coordinate, size * sizeof(float), hipMemcpyHostToDevice);

	//call kernel function
	positionCalculateExecution << <1, size >> > (dev_coordinate, dev_dis, dev_rPos);

	//Copy output arry from GPU device to Host
	hipMemcpy(relativePos, dev_rPos, size * sizeof(float), hipMemcpyDeviceToHost);// Not Working 

	//Free device memory
	hipFree(dev_coordinate);
	hipFree(dev_dis);
	hipFree(dev_rPos);
}


